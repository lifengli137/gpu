#include <stdio.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#define CRITICALTHRESHOLD	3
#define WARNINGTHRESHOLD	6
#define CRITICAL	2
#define WARNING		1
#define OK			0
#define MB 1024*1024

char* checkCuda(hipError_t result){
	if (result != hipSuccess) {
		printf("CUDA Runtime Error: %s\n", hipGetErrorString(result));
	}
	return 0;
}

int main(){
	int res = OK;
	

	int devices = 0;
	checkCuda(hipGetDeviceCount(&devices));
		
	char pcibus[32];
	int bytes = 1024 * MB;
	int tests = 3;
	for(int i = 0; i < devices; i++){
		hipDeviceProp_t prop;
		checkCuda( hipGetDeviceProperties(&prop, 0) );
		if(strstr(prop.name, "M40") != NULL || strstr(prop.name, "K40") != NULL){
			printf("Does not support %s.\n", prop.name);
			continue;
		}
	}
	
	for(int i = 0; i < devices; i++){
		for(int j = 0; j < devices; j++){
			if(i == j) continue;

			int accessible_i_j = 0;
			checkCuda(hipDeviceCanAccessPeer(&accessible_i_j, i, j));
			if(!accessible_i_j){
				printf("GPU %d is not able to P2P access to GPU %d.\n", i, j);
				continue;
			}
			
			int accessible_j_i = 0;
			checkCuda(hipDeviceCanAccessPeer(&accessible_j_i, j, i));
			if(!accessible_j_i){
				printf("GPU %d is not able to P2P access to GPU %d.\n", j, i);
				continue;
			}
			double i_to_j = 0;
			double j_to_i = 0;
			checkCuda(hipSetDevice(i));
			hipDeviceEnablePeerAccess(j, 0);
			checkCuda(hipSetDevice(j));
			hipDeviceEnablePeerAccess(i, 0);
			char* d_mem_i;
			char* d_mem_j;
			checkCuda(hipSetDevice(i));
			checkCuda(hipMalloc( (void**)&d_mem_i, sizeof(char)*bytes));
			checkCuda(hipSetDevice(j));
			checkCuda(hipMalloc( (void**)&d_mem_j, sizeof(char)*bytes));
			hipEvent_t start, stop;
			checkCuda(hipEventCreate(&start));
			checkCuda(hipEventCreate(&stop));
			checkCuda(hipEventRecord(start, 0));
			for(int j = 0; j < tests; j++)
				checkCuda(hipMemcpy( d_mem_i, d_mem_j, sizeof(char)*bytes, hipMemcpyDefault ));
			checkCuda(hipEventRecord(stop, 0));
			checkCuda(hipEventSynchronize(stop));
			float t;
			checkCuda(hipEventElapsedTime(&t, start, stop));
			i_to_j = (double)bytes*1e-6/(t/tests);
			checkCuda(hipEventRecord(start, 0));
			for(int j = 0; j < tests; j++)
				checkCuda(hipMemcpy( d_mem_j, d_mem_i, sizeof(char)*bytes, hipMemcpyDefault ));
			checkCuda(hipEventRecord(stop, 0));
			checkCuda(hipEventSynchronize(stop));
			checkCuda(hipEventElapsedTime(&t, start, stop));
			j_to_i = (double)bytes*1e-6/(t/tests);
			printf("GPU %d P2P GPU %d: %2.2fGB/s  GPU %d P2P GPU %d: %2.2fGB/s\n", i, j, i_to_j, j, i, j_to_i);
			checkCuda(hipSetDevice(i));
			checkCuda(hipFree(d_mem_i));
			checkCuda(hipSetDevice(j));
			checkCuda(hipFree(d_mem_j));
			checkCuda( hipEventDestroy(start) );
			checkCuda( hipEventDestroy(stop) );
		}
		
	}
	
	
	return res;
}
